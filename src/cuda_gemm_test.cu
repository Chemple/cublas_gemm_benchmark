#include <chrono>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <memory>
#include <sstream>
#include <stdexcept>
#include <tuple>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "tensor.h"

#ifndef PAD_KERNELS
#define PAD_KERNELS 1
#endif

template <typename T1, typename T2>
auto lauch_cublas_gemm(Tensor<T1> A, Tensor<T1> B, Tensor<T2> C, bool a_t,
                       bool b_t, hipblasHandle_t cublas_handle,
                       bool use_tensor_core) {

  const int alpha = 1.f;
  const int beta = 1.f;

  int m = C.dims()[0];
  int k = a_t ? A.dims()[0] : A.dims()[1];
  int n = C.dims()[1];

  hipblasStatus_t stat;

  hipDataType A_type = HIP_R_16F;
  hipDataType B_type = HIP_R_16F;
  hipDataType C_type = HIP_R_32F;
  hipDataType compute_type = HIP_R_32F;
  hipblasGemmAlgo_t algo;

  algo = use_tensor_core ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT;

  stat =
      hipblasGemmEx(cublas_handle, a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                   b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &alpha, A.begin(),
                   A_type, A.dims()[0], B.begin(), B_type, B.dims()[0], &beta,
                   C.begin(), C_type, C.dims()[0], compute_type, algo);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    throw std::runtime_error("sgemm failed");
  }

  hipDeviceSynchronize();
}

int main() {
  hipblasHandle_t cublas_handle;
  hipblasStatus_t status = hipblasCreate(&cublas_handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cout << "CUBLAS init failed" << std::endl;
  }

  int m = 32;
  int n = 512;
  int k = 1024;

  status = hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cout << "CUBLAS math mode failed" << std::endl;
  }

  {
    
  }
}